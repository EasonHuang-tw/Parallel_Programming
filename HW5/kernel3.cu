#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
/*
__global__ static inline int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}
*/
__global__ void mandelKernel( int* d_data,float stepX, float stepY,float lowerX,float lowerY ,int width, int maxIterations, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
	int thisX= blockIdx.x * blockDim.x + threadIdx.x;
	int thisY= blockIdx.y * blockDim.y + threadIdx.y;
	if( (thisX % 16 == 0) && (thisY % 16 == 0)  ){
		int j = thisX , k = thisY;
		for(j=thisX;j < thisX+16;j++){
			for( k=thisY ; k < thisY+16 ; k++){
				float x = lowerX + j * stepX;
				float y = lowerY + k * stepY;
				  
				float c_re = x, c_im = y;
				float z_re = c_re, z_im = c_im;

				int i;
				for (i = 0; i < maxIterations; ++i)
				{

						if (z_re * z_re + z_im * z_im > 4.f)
						break;

						float new_re = z_re * z_re - z_im * z_im;
						float new_im = 2.f * z_re * z_im;
						z_re = c_re + new_re;
						z_im = c_im + new_im;
				}
				int *ptr = (int *)((char*)d_data+k*pitch);
				ptr[j] = i;
				//d_data[ thisX + thisY * width ] = i;
			}
		}
	}
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
	int *data, *d_data;
	dim3 threadPerBlock(25,25);
	dim3 numBlocks(resX/threadPerBlock.x,resY/threadPerBlock.y);
	size_t pitch;
	//data = (int*)malloc( sizeof(int)*resX*resY );
	hipHostAlloc(&data, sizeof(int) * resX*resY, hipHostMallocMapped);

	hipMallocPitch((void **)&d_data, &pitch, sizeof(int)*resX, resY);
//	cudaMalloc((void**)&d_data, sizeof(int)*resX*resY );

	mandelKernel<<<numBlocks,threadPerBlock>>>(d_data,stepX,stepY,lowerX,lowerY,resX,maxIterations,pitch);

 	hipMemcpy2D(data, sizeof(int)*resX, d_data, pitch, sizeof(int)*resX, resY, hipMemcpyDeviceToHost);

	memcpy(img,data,sizeof(int)*resX*resY);
	hipHostFree(data);
	hipFree(d_data);
	
}
